#include "hip/hip_runtime.h"
#ifndef __CUDA_ARCH__DEVICE_HOST__
#define __CUDA_ARCH__DEVICE_HOST__ 1
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>

#include <iostream>
#include <stdint.h>
#include <math.h>
#include <string>

#include <CUDASieve/cudasieve.hpp>
#include <CUDASieve/host.hpp>

#include "cuda_uint128.h"
#include "cuda_uint128_primitives.cuh"



uint128_t calc(char * argv);


uint64_t * generateUniform64(uint64_t num);

__global__
void atimesbequalsc(uint64_t * a, uint64_t * b, uint128_t * c);
__global__
void squarerootc(uint128_t * c, uint64_t * a);
__global__
void sqrt_test(uint64_t * a, volatile uint64_t * errors);
__global__
void div_test(uint64_t * a, volatile uint64_t * errors);

uint128_t calc(char* argv) // for getting values bigger than the 32 bits that system() will return;
{
    uint128_t value;
    size_t len = 0;
    char* line = NULL;
    FILE* in;
    char cmd[256];

#if 0
    sprintf(cmd, "calc %s | awk {'print $1'}", argv);
    in = popen(cmd, "r");
    getline(&line, &len, in);
    std::string s = line;
    value = string_to_u128(s);
#endif

    return value;
}

void all_test(int argc, char* argv[]) {

    uint128_t x = 0;
    if (argc == 2) {
        x = calc(argv[1]);
    }

    size_t len;

    uint64_t* d_primes = CudaSieve::getDevicePrimes(0, pow(10, 9), len, 0);

    x = cuda128::reduce64to128(d_primes, x.lo);
    std::cout << x << std::endl;


    // uint64_t * d64 = generateUniform64(1u<<26);
    // volatile uint64_t * h_errors, * d_errors;
    // hipHostAlloc((void **)&h_e rrors, sizeof(uint64_t), hipHostMallocMapped);
    // hipHostGetDevicePointer((uint64_t **)&d_errors, (uint64_t *)h_errors, 0);
    //
    // *h_errors = 0;
    //
    // KernelTime timer;
    //
    // timer.start();
    //
    // div_test<<<65536, 256>>>(d64, d_errors);
    //
    // hipDeviceSynchronize();
    // timer.stop();
    // timer.displayTime();
    //
    // std::cout << *h_errors << " errors " << std::endl;

}

int main(int argc, char* argv[])
{
    all_test(argc, argv);
    return 0;
}



uint64_t * generateUniform64(uint64_t num)
{
  uint64_t * d_r;
  hiprandGenerator_t gen;

  hipMalloc(&d_r, num * sizeof(uint64_t));

  hiprandCreateGenerator(&gen, HIPRAND_RNG_QUASI_SOBOL64);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1278459ull);
  hiprandGenerateLongLong(gen, (unsigned long long *)d_r, num);

  return d_r;
}

__global__
void atimesbequalsc(uint64_t * a, uint64_t * b, uint128_t * c)
{
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  c[tidx] = mul128(a[tidx], b[tidx]);
}

__global__
void squarerootc(uint128_t * c, uint64_t * a)
{
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  a[tidx] = _isqrt(c[tidx]);
  if(mul128(a[tidx], a[tidx]) > c[tidx] || mul128((a[tidx] + 1), (a[tidx] + 1)) <= c[tidx])
    printf("%llu  %f  %llu\n", a[tidx], u128_to_float(c[tidx]), c[tidx].hi);
}

__global__
void sqrt_test(uint64_t * a, volatile uint64_t * errors)
{
  __shared__ uint64_t s_a[1024];

  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    s_a[threadIdx.x + i * blockDim.x] = a[threadIdx.x + i * blockDim.x + 1024*blockIdx.x];
  }
  __syncthreads();

  uint128_t x;
  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    x.lo = s_a[threadIdx.x + i * blockDim.x];
    #pragma unroll
    for(uint16_t i = 0; i < 1024; i++){
      x.hi = s_a[i] >> 4;
      uint64_t y = _isqrt(x);
      if(mul128(y,y) > x || mul128(y + 1, y + 1) <= x){
        atomicAdd((unsigned long long *)errors, 1ull);
        printf("%llu %llu %llu\n", x.hi, x.lo, y);
      }
    }
  }
}

__global__
void div_test(uint64_t * a, volatile uint64_t * errors)
{
  __shared__ uint64_t s_a[1024];

  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    s_a[threadIdx.x + i * blockDim.x] = a[threadIdx.x + i * blockDim.x + 1024*blockIdx.x];
  }
  __syncthreads();

  uint128_t x, y;
  uint64_t v, r;
  #pragma unroll
  for(uint16_t i = 0; i < 4; i++){
    x.lo = s_a[threadIdx.x + i * blockDim.x];
    #pragma unroll
    for(uint16_t i = 0; i < 1024; i++){
      x.hi = s_a[i] >> 4;
      v = s_a[(i + 1 )& 1023] >> (x.hi & 31);
      y = div128to128(x,v,&r);
      y = add128(mul128(y, v), r);
      uint64_t y = _isqrt(x);
      // if(y != x){
      //   atomicAdd((unsigned long long *)errors, 1ull);
      // }
    }
  }
}

